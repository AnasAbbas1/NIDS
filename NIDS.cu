#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipcub/hipcub.hpp"
#include "cub/device/device_scan.cuh"
#include <stdio.h>
#include <chrono>
#include<iostream>
#include<vector>
#include<algorithm>
#include<string>
#include <fstream>
#include<set>
#include <stdlib.h>
#include <time.h>
#include <random>

using namespace std;
using namespace std::chrono;
using namespace hipcub;
CachingDeviceAllocator g_allocator(true);
#define ull unsigned long long
const ull h_q = 65521;
const int h_p = 1024;
const ull h_n = 1 << 20;
const ull h_m = 13;
const ull h_mps[] = {7, 31, 127, 8191, 131071, 524287};
const int h_masksz = 6;
const ull h_shifts[] = {3, 5, 7, 13, 17, 19};
const ull h_cumShifts[] = {0, 3, 8, 15, 28, 45};
const ull h_ds[] = {2, 3, 5, 11, 13, 17};
const ull h_HTSZ = 1 << 18;
const int h_d = 2;
__constant__ const int d_d = 2;
__constant__ const ull d_q = 65521;
__constant__ const ull d_n = 1 << 20;
__constant__ const ull d_m = 13;
__constant__ const ull d_mps[] = {7, 31, 127, 8191, 131071, 524287};
__constant__ const int d_masksz = 6;
__constant__ const ull d_masks[] = {7, 248, 32512, 268402688, 35184103653376ull, 18446708889337462784ull};
__constant__ const ull d_nmasks[] = {18446744073709551608ull, 18446744073709551367ull, 18446744073709519103ull, 18446744073441148927ull, 18446708889605898239ull, 35184372088831ull};
__constant__ const ull d_shifts[] = {3, 5, 7, 13, 17, 19};
__constant__ const ull d_cumShifts[] = {0, 3, 8, 15, 28, 45};
__constant__ const ull d_ds[] = {2, 3, 5, 11, 13, 17};
__constant__ const ull d_HTMSK = h_HTSZ - 1;
char* g_h_data;
char* g_h_patterns;
char* g_d_data;
char* g_d_patterns;
struct testcase {
private:
    string input_str;
    vector<pair<int, int>>expectedMatches;
    char* StringGeneration(int sz) {
        char* ret = new char[sz + 1];
        mt19937 generator{ random_device{}() };
        uniform_int_distribution<int> distribution{ 'a', 'a' + h_d - 1 };
        string rand_str(sz, '\0');
        for (auto& dis : rand_str)
            dis = distribution(generator);
        for (int i = 0; i < sz; i++) {
            ret[i] = rand_str[i];
        }
        ret[sz] = 0;
        return ret;
    }
    void WriteData(char* data) {
        ofstream myfile;
        myfile.open("outputfiles\\data.txt");
        myfile << string(data);
        myfile.close();
    }
    void WritePatterns(char * patterns) {
        set<string>st;
        ofstream myfile;
        myfile.open("outputfiles\\patterns.txt");
        for (int patternIndex = 0; patternIndex < h_p; patternIndex++) {
            string pattern = "";
            for (int i = patternIndex * h_m; i < patternIndex * h_m + h_m; i++)
                pattern += g_h_patterns[i];
            st.insert(pattern);
            myfile << patternIndex << ": " << pattern << endl;
        }
        if (st.size() != h_p) {
            cout << "Duplicate pattern occurred" << endl; 
        }
        myfile.close();
    }
    void WriteMatches(vector<pair<int, int>> matches, string fileName) {
        ofstream myfile;
        myfile.open(fileName.c_str());
        sort(matches.begin(), matches.end());
        myfile << "Row#\tPattern Index\tposition" << endl;
        for (int i = 0; i < matches.size(); i++) {
            myfile << i << ":\t"<<matches[i].first << "\t\t" << matches[i].second << endl;
        }
        myfile.close();
    }
    char * PatternsGeneration(){
        set<string>st;
        while(st.size() != h_p){
            char * ptrn = StringGeneration(h_m);
            st.insert(string(ptrn));
        }
        char * ret = new char[h_p * h_m + 1];
        ret[h_p * h_m] = 0;
        while(st.size()){
            for(int i = 0; i < h_m; i++){
                ret[h_m * (h_p - st.size()) + i] = (*st.begin())[i];
            }
            st.erase(st.begin());
        }
        return ret;
    }
    void GenerateInputData() {
        g_h_data = StringGeneration(h_n);
        g_d_data = NULL;
        g_h_patterns = PatternsGeneration();
        g_d_patterns = NULL;
    }
    void FindPattern(int patternIndex) {
        string pattern = "";
        for (int i = patternIndex * h_m; i < patternIndex * h_m + h_m; i++)
            pattern += g_h_patterns[i];

        size_t pos = input_str.find(pattern);
        while (pos != string::npos) {
            expectedMatches.push_back({ patternIndex, pos });
            pos = input_str.find(pattern, pos + 1);
        }

    }
    void SolveOnCPU() {
        input_str = string(g_h_data);
        for (int i = 0; i < h_p; i++) 
            FindPattern(i);
    }
public:
    testcase() {
        GenerateInputData();
        WriteData(g_h_data);
        WritePatterns(g_h_patterns);
        SolveOnCPU();
        WriteMatches(expectedMatches, "outputfiles\\Expected.txt");
    }
    static void CopyDataToDevice(){
        CubDebugExit(g_allocator.DeviceAllocate((void**)&g_d_data, sizeof(char) * h_n));
        CubDebugExit(hipMemcpy(g_d_data, g_h_data, sizeof(char) * h_n, hipMemcpyHostToDevice));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&g_d_patterns, sizeof(char) * h_p * h_m));
        CubDebugExit(hipMemcpy(g_d_patterns, g_h_patterns, sizeof(char) * h_p * h_m, hipMemcpyHostToDevice));
        // debug
        //delete[] g_h_patterns;
        //delete[] g_h_data;
    }
    void Validate(int* h_output) {
        vector<pair<int, int>>gpuMatches;
        for (int i = 0; i < h_n; i++) {
            if (h_output[i] != -1) {
                gpuMatches.push_back({h_output[i], i});
            }
        }
        sort(gpuMatches.begin(), gpuMatches.end());
        WriteMatches(gpuMatches, "outputfiles\\Actual.txt");
        bool same = true;
        if (gpuMatches.size() != expectedMatches.size()) {
            cout << "sizes are not equal expected size is " << expectedMatches.size() << " and actual size is " << gpuMatches.size() << endl;
            same = false;
        }
        for (int i = 0, limit = 0;i < min(gpuMatches.size(), expectedMatches.size()); i++) {
            if (gpuMatches[i].first != expectedMatches[i].first || gpuMatches[i].second != expectedMatches[i].second) {
                cout << "Mismatch at position: " << i << endl;
                limit++;
                same = false;
                if(limit >= 100){
                    break;
                }
            }
        }
        if (same) {
            cout << "Code works fine" << endl;
        }
        else {
            cout << "Results doesn't match, debug your code" << endl;
        }
    }

}test;
struct CustomSum
{
    HIPCUB_RUNTIME_FUNCTION __host__ __device__ __forceinline__
        int operator()(const int& a, const int& b) const {
        return (a + b) % d_q;
    }
}sumMod;
struct CustomSumNew
{
    HIPCUB_RUNTIME_FUNCTION __device__ __forceinline__
        ull operator()(const ull& a, const ull& b) const {
            ull ans = 0;
            for(int j = 0; j < d_masksz; j++){
                ull sum = ((a & d_masks[j]) >> d_cumShifts[j]) + ((b & d_masks[j]) >> d_cumShifts[j]);
                sum = (sum & d_mps[j]) + (sum >> d_shifts[j]);
                sum = sum >= d_mps[j] ? sum - d_mps[j] : sum;
                ans |= sum << d_cumShifts[j];
            }
        return ans;
    }
}sumModMersennePrime;
__global__ void CalculateHashPattern(char* d_patterns, int* d_controlArray, int* d_hashTable) {
    int patternIndex = threadIdx.x, patternHash = 0;

    for (int i = patternIndex * d_m; i < patternIndex * d_m + d_m; i++)
        patternHash = (patternHash * d_d + (d_patterns[i] - 'a' + 1)) % d_q;

    while (atomicAdd(&d_controlArray[patternHash], 1) != 0)
        patternHash = (patternHash + 1) % d_q;

    d_hashTable[patternHash] = patternIndex;
}
__global__ void CalculateHashPatternNew(char* d_patterns, int* d_controlArray, int* d_hashTable, ull* d_patternHashes) {
    int patternIndex = threadIdx.x;
    ull patternHash = 0;

    for (int i = patternIndex * d_m; i < patternIndex * d_m + d_m; i++){
        for(int j = 0; j < d_masksz; j++){
            ull hash = (patternHash & d_masks[j]) >> d_cumShifts[j];
            hash = hash * d_ds[j] + (ull)(d_patterns[i] - 'a' + 1);
            hash = (hash & d_mps[j]) + (hash >> d_shifts[j]);
            hash = hash >= d_mps[j] ? hash - d_mps[j] : hash;
            patternHash &= d_nmasks[j];
            patternHash |= hash << d_cumShifts[j];
        }
    }

    d_patternHashes[patternIndex] = patternHash;
    while (atomicAdd(&d_controlArray[patternHash & d_HTMSK], 1) != 0)
        patternHash = (patternHash == d_HTMSK) ? 0: patternHash + 1;

    d_hashTable[patternHash & d_HTMSK] = patternIndex;
}
__global__ void CalculateHashes(int* d_a, char* d_data, int* d_lookupTable) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[i] = (d_lookupTable[(d_n - i - 1) % (d_q - 1)] * (d_data[i] - 'a' + 1)) % d_q;
}
__global__ void CalculateHashesNew(ull* d_a, char* d_data, ull* d_lookupTable) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[i] = 0;
    for(int j = 0; j < d_masksz; j++){
        ull hash = (((d_lookupTable[(d_n - i - 1) % (d_mps[j] - 1)] & d_masks[j]) >> d_cumShifts[j]) * (ull)(d_data[i] - 'a' + 1)); //hash =  (ds[j]^(i mod (mps[j] - 1)) % mps[j]) * data[i]
        hash = (hash & d_mps[j]) + (hash >> d_shifts[j]);
        hash = hash >= d_mps[j] ? hash - d_mps[j] : hash;
        d_a[i] |= hash << d_cumShifts[j];
    }
    
}
__global__ void FindMatches(int* d_prefixSum, char* d_data, char* d_patterns, int* d_lookupTable, int* d_controlArray, int* d_hashTable, int* d_output) {
    ull j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j + d_m - 1 <= d_n) {
        int hash = ((((ull)(d_prefixSum[j + d_m - 1] - (j ? d_prefixSum[j - 1] : 0)) + d_q) % d_q) * (ull)d_lookupTable[(d_m + ((d_n - j + d_q - 2ll) / (d_q - 1ll)) * (d_q - 1ll) - d_n + j) % (d_q - 1ll)]) % d_q;
        while (d_controlArray[hash]) {
            int patternIndex = d_hashTable[hash];
            bool match = true;
            for (int i = patternIndex * d_m, offset = 0; i < patternIndex * d_m + d_m; i++, offset++) {
                if (d_patterns[i] != d_data[j + offset]) {
                    match = false;
                    break;
                }
            }
            if (match) {
                d_output[j] = patternIndex;
                return; 
            }
            hash = (hash + 1) % d_q;
        }
    }
}
__global__ void FindMatchesNew(ull* d_prefixSum, ull* d_lookupTable, int* d_controlArray, int* d_hashTable, int* d_output, ull* d_patternHashes) {
    ull j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j + d_m - 1 <= d_n) {
        ull hash = 0;
        for(int k = 0; k < d_masksz; k++){
            ull tmp = (d_prefixSum[j + d_m - 1] & d_masks[k]) >> d_cumShifts[k];
            if(j){
                tmp = (tmp + d_mps[k]) - ((d_prefixSum[j - 1] & d_masks[k]) >> d_cumShifts[k]);
                tmp = (tmp >= d_mps[k] ? tmp - d_mps[k] : tmp);
            }
            tmp = tmp * ((d_lookupTable[(d_m + ((d_n - j + d_mps[k] - 2ll) / (d_mps[k] - 1ll)) * (d_mps[k] - 1ll) - d_n + j) % (d_mps[k] - 1ll)] & d_masks[k]) >> d_cumShifts[k]) ;
            tmp = (tmp & d_mps[k]) + (tmp >> d_shifts[k]);
            hash |= (tmp >= d_mps[k] ? tmp - d_mps[k] : tmp) << d_cumShifts[k]; 
        }
        for(int i = 0; i < d_p; i++){
            if (hash == d_patternHashes[i]) {
                d_output[j] = i;
                return;
            }
        }
        /*
        while (d_controlArray[hash & d_HTMSK]) {
            if (hash == d_patternHashes[d_hashTable[hash & d_HTMSK]]) {
                d_output[j] = d_hashTable[hash & d_HTMSK];
                return;
            }
            hash = (hash == d_HTMSK) ? 0: hash + 1;
        }
        */
    }
}
class PaperImplementation{
private:
    static int* Step1() {
        int* ret = NULL;
        int *h_lookupTabe = new int [h_q];
        for (int i = 0, current = 1; i < h_q; i++, current = (current * h_d) % h_q ) 
            h_lookupTabe[i] = current;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&ret, sizeof(int) * h_q));
        CubDebugExit(hipMemcpy(ret, h_lookupTabe, sizeof(int) * h_q, hipMemcpyHostToDevice));
        delete[] h_lookupTabe;
        hipDeviceSynchronize();
        return ret;
    }
    static pair<int*, int*> Step2(char * d_patterns) {
        int* d_controlArray = NULL, * d_hashTable = NULL, * h_controlArray = new int[h_q], * h_hashTable = new int[h_q];
        for (int i = 0; i < h_q; i++) {
            h_controlArray[i] = 0;
            h_hashTable[i] = -1;
        }
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_controlArray, sizeof(int) * h_q));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_hashTable, sizeof(int) * h_q));
        CubDebugExit(hipMemcpy(d_controlArray, h_controlArray, sizeof(int) * h_q, hipMemcpyHostToDevice));
        CubDebugExit(hipMemcpy(d_hashTable, h_hashTable, sizeof(int) * h_q, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        CalculateHashPattern <<< 1, h_p >>> (d_patterns, d_controlArray, d_hashTable);
        hipDeviceSynchronize();
        delete[] h_hashTable;
        delete[] h_controlArray;
        return { d_controlArray, d_hashTable };
    }
    static int* Step3(char * d_data, int* d_lookupTable) {
        int* d_a = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_a, sizeof(int) * h_n));
        CalculateHashes << <h_n / 256, 256 >> > (d_a, d_data, d_lookupTable);
        hipDeviceSynchronize();
        return d_a;
    }
    static int* Step4(int* d_a) {
        int* d_prefixSum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_prefixSum, sizeof(int) * h_n));
        hipDeviceSynchronize();
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumMod, h_n));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumMod, h_n));
        hipFree(d_a);
        hipFree(d_temp_storage);
        return d_prefixSum;
    }
    static int* Step5(int* d_prefixSum, char* d_data, char* d_patterns, int* d_lookupTable, int* d_controlArray, int* d_hashTable) {
        int* h_output = new int [h_n];
        for (int i = 0; i < h_n; i++)
            h_output[i] = -1;
        int* d_output = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(int) * h_n));
        CubDebugExit(hipMemcpy(d_output, h_output, sizeof(int) * h_n, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        FindMatches << <h_n / 256, 256 >> > (d_prefixSum, d_data, d_patterns, d_lookupTable, d_controlArray, d_hashTable, d_output);
        CubDebugExit(hipMemcpy(h_output, d_output, sizeof(int) * h_n, hipMemcpyDeviceToHost));
        hipFree(d_output);
        hipFree(d_prefixSum);
        hipFree(d_data);
        hipFree(d_patterns);
        hipFree(d_lookupTable);
        hipFree(d_controlArray);
        hipFree(d_hashTable);
        return h_output;
    }
public:
    static int* Execute() {
        //CopyDataToDevice();
        //1.Load a preprocessed lookup table for di mod q (0 ≤ i ≤ q − 1)
        int* d_lookupTable = Step1();
        //2. Compute the values of h(Pk) for all k (0 ≤ k ≤ p − 1) in parallel and create the hash table HT using the calculated values.
        pair<int*, int*> p = Step2(g_d_patterns);
        //3.Compute the a0, a1,..., an−1 in parallel.
        int* d_a = Step3(g_d_data, d_lookupTable);
        //4.Compute the prefix-sums ˆa0, aˆ1,..., aˆn−1.
        int* d_prefixSum = Step4(d_a);
        //5.  For all j (0 ≤ j ≤ n − m), compute ( ˆaj+m−1 − aˆ j−1) · dm−n−j, which is equal to h(tjtj + 1 ... tj + m−1).If array control[h(tjtj + 1 ... tj + m−1)]  0 then compare the characters of text and pattern with Match(i, j).
        int* h_output = Step5(d_prefixSum, g_d_data, g_d_patterns, d_lookupTable, p.first, p.second);
    
        return h_output;
    }
};
class ProposedImplementation{
private:
    static ull* Step1(){
        ull* ret = NULL;
        ull* h_lookupTabe = new ull [h_mps[5]];
        ull currents[] = {1, 1, 1, 1, 1, 1};
        for(int i = 0; i < h_mps[5]; i++){
            h_lookupTabe[i] = 0;
            for(int j = 0; j <h_masksz; j++){
                h_lookupTabe[i] |= currents[j] << h_cumShifts[j];
                currents[j] = currents[j] * h_ds[j];
                currents[j] = (currents[j] & h_mps[j]) + (currents[j] >> h_shifts[j]);
                currents[j] = currents[j] >= h_mps[j] ? currents[j] - h_mps[j] : currents[j];
            }
            
        }    
        CubDebugExit(g_allocator.DeviceAllocate((void**)&ret, sizeof(ull) * h_mps[5]));
        CubDebugExit(hipMemcpy(ret, h_lookupTabe, sizeof(ull) * h_mps[5], hipMemcpyHostToDevice));
        delete[] h_lookupTabe;
        hipDeviceSynchronize();
        return ret;
    }
    static pair<pair<int*, int*>, ull*> Step2(char * d_patterns) {
        int* d_controlArray = NULL, * d_hashTable = NULL,* h_controlArray = new int[h_HTSZ],* h_hashTable = new int [h_HTSZ];
        ull * d_patternHashes = NULL;
        for (int i = 0; i < h_HTSZ; i++) {
            h_controlArray[i] = 0;
            h_hashTable[i] = -1;
        }
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_patternHashes, sizeof(ull) * h_p));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_controlArray, sizeof(int) * h_HTSZ));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_hashTable, sizeof(int) * h_HTSZ));
        CubDebugExit(hipMemcpy(d_controlArray, h_controlArray, sizeof(int) * h_HTSZ, hipMemcpyHostToDevice));
        CubDebugExit(hipMemcpy(d_hashTable, h_hashTable, sizeof(int) * h_HTSZ, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        CalculateHashPatternNew <<< 1, h_p >>> (d_patterns, d_controlArray, d_hashTable, d_patternHashes);
        hipFree(d_patterns);
        hipDeviceSynchronize();
        delete[] h_hashTable;
        delete[] h_controlArray;
        return { {d_controlArray, d_hashTable}, d_patternHashes};
    }
    static ull* Step3(char * d_data, ull* d_lookupTable) {
        ull* d_a = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_a, sizeof(ull) * h_n));
        CalculateHashesNew << <h_n / 256, 256 >> > (d_a, d_data, d_lookupTable);
        hipFree(d_data);
        hipDeviceSynchronize();
        return d_a;
    }
    static ull* Step4(ull* d_a) {
        ull* d_prefixSum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_prefixSum, sizeof(ull) * h_n));
        hipDeviceSynchronize();
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumModMersennePrime, h_n));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumModMersennePrime,h_n));
        hipFree(d_a);
        hipFree(d_temp_storage);
        return d_prefixSum;
    }
    static int* Step5(ull* d_prefixSum, ull* d_lookupTable, int* d_controlArray, int* d_hashTable, ull* d_patternHashes) {
        int* h_output = new int [h_n];
        for (int i = 0; i < h_n; i++)
            h_output[i] = -1;
        int* d_output = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(int) * h_n));
        CubDebugExit(hipMemcpy(d_output, h_output, sizeof(int) * h_n, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        FindMatchesNew << <h_n / 256, 256 >> > (d_prefixSum, d_lookupTable, d_controlArray, d_hashTable, d_output, d_patternHashes);
        CubDebugExit(hipMemcpy(h_output, d_output, sizeof(int) * h_n, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        hipFree(d_output);
        hipFree(d_prefixSum);
        hipFree(d_lookupTable);
        hipFree(d_controlArray);
        hipFree(d_hashTable);
        hipFree(d_patternHashes);
        return h_output;
    }
public:
    static int* Execute() {
        ull* d_lookupTable = Step1();
        cout << "step 1 done" << endl;
        static pair<pair<int*, int*>, ull*> p = Step2(g_d_patterns);
        cout << "step 2 done" << endl;
        ull* d_a = Step3(g_d_data, d_lookupTable);
        cout << "step 3 done" << endl;
        ull* d_prefixSum = Step4(d_a);
        cout << "step 4 done" << endl;
        int* h_output = Step5(d_prefixSum, d_lookupTable, p.first.first, p.first.second, p.second);
        cout << "step 5 done" << endl;
        return h_output;
    }
};
int main(){
    testcase::CopyDataToDevice();
    test.Validate(ProposedImplementation::Execute());
    return 0;
}
