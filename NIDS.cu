#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipcub/hipcub.hpp"
#include "cub/device/device_scan.cuh"
#include <stdio.h>
#include <chrono>
#include<iostream>
#include<vector>
#include<algorithm>
#include<string>
#include <fstream>
#include<set>
#include <stdlib.h>
#include <time.h>
#include <random>

using namespace std;
using namespace std::chrono;
using namespace hipcub;
CachingDeviceAllocator g_allocator(true);
#define ull unsigned long long
__constant__ const int d_d = 2;
__constant__ const ull d_q = 65521;
__constant__ const int d_p = 1024;
__constant__ const ull d_n = 1 << 18;
__constant__ const ull d_m = 13;
__constant__ const ull d_mps[] = {7, 31, 127, 8191, 131071, 524287};
__constant__ const int d_masksz = 6;
__constant__ const ull d_masks[] = {7, 248, 32512, 268402688, 35184103653376ull, 18446708889337462784ull};
__constant__ const ull d_nmasks[] = {18446744073709551608ull, 18446744073709551367ull, 18446744073709519103ull, 18446744073441148927ull, 18446708889605898239ull, 35184372088831ull};
__constant__ const ull d_shifts[] = {3, 5, 7, 13, 17, 19};
__constant__ const ull d_cumShifts[] = {0, 3, 8, 15, 28, 45};
__constant__ const ull d_ds[] = {2, 3, 5, 11, 13, 17};
__constant__ const ull d_HTSZ = 1 << 18;
__constant__ const ull d_HTMSK = HTSZ - 1;
const int h_d = 2;
const ull h_q = 65521;
const int h_p = 1024;
const ull h_n = 1 << 18;
const ull h_m = 13;
const ull h_mps[] = {7, 31, 127, 8191, 131071, 524287};
const int h_masksz = 6;
const ull h_masks[] = {7, 248, 32512, 268402688, 35184103653376ull, 18446708889337462784ull};
const ull h_nmasks[] = {18446744073709551608ull, 18446744073709551367ull, 18446744073709519103ull, 18446744073441148927ull, 18446708889605898239ull, 35184372088831ull};
const ull h_shifts[] = {3, 5, 7, 13, 17, 19};
const ull h_cumShifts[] = {0, 3, 8, 15, 28, 45};
const ull h_ds[] = {2, 3, 5, 11, 13, 17};
const ull h_HTSZ = 1 << 18;
const ull h_HTMSK = HTSZ - 1;
struct testcase {
private:
    string input_str;
    vector<pair<int, int>>expectedMatches;
    char* StringGeneration(int sz) {
        char* ret = new char[sz + 1];
        mt19937 generator{ random_device{}() };
        uniform_int_distribution<int> distribution{ 'a', 'a' + h_d - 1 };
        string rand_str(sz, '\0');
        for (auto& dis : rand_str)
            dis = distribution(generator);
        for (int i = 0; i < sz; i++) {
            ret[i] = rand_str[i];
        }
        ret[sz] = 0;
        return ret;
    }
    void WriteData(char* data) {
        ofstream myfile;
        myfile.open("outputfiles\\data.txt");
        myfile << string(data);
        myfile.close();
    }
    void WritePatterns(char * patterns) {
        set<string>st;
        ofstream myfile;
        myfile.open("outputfiles\\patterns.txt");
        for (int patternIndex = 0; patternIndex < p; patternIndex++) {
            string pattern = "";
            for (int i = patternIndex * m; i < patternIndex * m + m; i++)
                pattern += h_patterns[i];
            st.insert(pattern);
            myfile << patternIndex << ": " << pattern << endl;
        }
        if (st.size() != p) {
            cout << "Duplicate pattern occurred" << endl; 
        }
        myfile.close();
    }
    void WriteMatches(vector<pair<int, int>> matches, string fileName) {
        ofstream myfile;
        myfile.open(fileName.c_str());
        sort(matches.begin(), matches.end());
        myfile << "Row#\tPattern Index\tposition" << endl;
        for (int i = 0; i < matches.size(); i++) {
            myfile << i << ":\t"<<matches[i].first << "\t\t" << matches[i].second << endl;
        }
        myfile.close();
    }
    char * PatternsGeneration(){
        set<string>st;
        while(st.size() != p){
            char * ptrn = StringGeneration(m);
            st.insert(string(ptrn));
        }
        char * ret = new char[p * m + 1];
        ret[p * m] = 0;
        while(st.size()){
            for(int i = 0; i < m; i++){
                ret[m * (p - st.size()) + i] = (*st.begin())[i];
            }
            st.erase(st.begin());
        }
        return ret;
    }
    void GenerateInputData() {
        h_data = StringGeneration(n);
        d_data = NULL;
        h_patterns = PatternsGeneration();
        d_patterns = NULL;
    }
    void FindPattern(int patternIndex) {
        string pattern = "";
        for (int i = patternIndex * m; i < patternIndex * m + m; i++)
            pattern += h_patterns[i];

        size_t pos = input_str.find(pattern);
        while (pos != string::npos) {
            expectedMatches.push_back({ patternIndex, pos });
            pos = input_str.find(pattern, pos + 1);
        }

    }
    void SolveOnCPU() {
        input_str = string(h_data);
        for (int i = 0; i < p; i++) 
            FindPattern(i);
    }
    void CopyDataToDevice(){
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(char) * n));
        CubDebugExit(hipMemcpy(d_data, h_data, sizeof(char) * n, hipMemcpyHostToDevice));
        CubDebugExit(g_allocator.DeviceAllocate((void**)d_patterns, sizeof(char) * p * m));
        CubDebugExit(hipMemcpy(d_patterns, h_patterns, sizeof(char) * p * m, hipMemcpyHostToDevice));
        delete[] h_patterns;
        delete[] h_data;
    }
public:
    char* h_data;
    char* h_patterns;
    char* d_data;
    char* d_patterns;
    testcase() {
        GenerateInputData();
        WriteData(h_data);
        WritePatterns(h_patterns);
        SolveOnCPU();
        WriteMatches(expectedMatches, "outputfiles\\Expected.txt");
        CopyDataToDevice();
    }

    void Validate(int* h_output) {
        vector<pair<int, int>>gpuMatches;
        for (int i = 0; i < n; i++) {
            if (h_output[i] != -1) {
                gpuMatches.push_back({h_output[i], i});
            }
        }
        sort(gpuMatches.begin(), gpuMatches.end());
        WriteMatches(gpuMatches, "outputfiles\\Actual.txt");
        bool same = true;
        if (gpuMatches.size() != expectedMatches.size()) {
            cout << "sizes are not equal expected size is " << expectedMatches.size() << " and actual size is " << gpuMatches.size() << endl;
            same = false;
        }
        for (int i = 0;i < min(gpuMatches.size(), expectedMatches.size()); i++) {
            if (gpuMatches[i].first != expectedMatches[i].first || gpuMatches[i].second != expectedMatches[i].second) {
                cout << "Mismatch at position: " << i << endl;
                same = false;
            }
        }
        if (same) {
            cout << "Code works fine" << endl;
        }
        else {
            cout << "Results doesn't match, debug your code" << endl;
        }
    }

}test;
struct CustomSum
{
    HIPCUB_RUNTIME_FUNCTION __host__ __device__ __forceinline__
        int operator()(const int& a, const int& b) const {
        return (a + b) % q;
    }
}sumMod;
struct CustomSumNew
{
    HIPCUB_RUNTIME_FUNCTION __host__ __device__ __forceinline__
        ull operator()(const ull& a, const ull& b) const {
            ull ans = 0;
            for(int j = 0; j < masksz; j++){
                ull sum = ((a & masks[j]) >> cumShifts[j]) + ((b & masks[j]) >> cumShifts[j]);
                sum = (sum & mps[j]) + (sum >> shifts[j]);
                sum = sum >= mps[j] ? sum - mps[j] : sum;
                ans |= sum << cumShifts[j];
            }
        return ans;
    }
}sumModMersennePrime;
__global__ void CalculateHashPattern(char* d_patterns, int* d_controlArray, int* d_hashTable) {
    int patternIndex = threadIdx.x, patternHash = 0;

    for (int i = patternIndex * m; i < patternIndex * m + m; i++)
        patternHash = (patternHash * d + (d_patterns[i] - 'a' + 1)) % q;

    while (atomicAdd(&d_controlArray[patternHash], 1) != 0)
        patternHash = (patternHash + 1) % q;

    d_hashTable[patternHash] = patternIndex;
}
__global__ void CalculateHashPatternNew(char* d_patterns, int* d_controlArray, int* d_hashTable, ull* d_patternHashes) {
    int patternIndex = threadIdx.x;
    ull patternHash = 0;

    for (int i = patternIndex * m; i < patternIndex * m + m; i++){
        for(int j = 0; j < masksz; j++){
            ull hash = (patternHash & masks[j]) >> cumShifts[j];
            hash = hash * ds[j] + (ull)(d_patterns[i] - 'a' + 1);
            hash = (hash & mps[j]) + (hash >> shifts[j]);
            hash = hash >= mps[j] ? hash - mps[j] : hash;
            patternHash &= nmasks[j];
            patternHash |= hash << cumShifts[j];
        }
    }

    d_patternHashes[patternIndex] = patternHash;
    while (atomicAdd(&d_controlArray[patternHash & HTMSK], 1) != 0)
        patternHash = (patternHash == HTMSK) ? 0: patternHash + 1;

    d_hashTable[patternHash & HTMSK] = patternIndex;
}
__global__ void CalculateHashes(int* d_a, char* d_data, int* d_lookupTable) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[i] = (d_lookupTable[(n - i - 1) % (q - 1)] * (d_data[i] - 'a' + 1)) % q;
}
__global__ void CalculateHashesNew(ull* d_a, char* d_data, ull* d_lookupTable) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[i] = 0;
    for(int j = 0; j < masksz; j++){
        ull hash = (((d_lookupTable[(n - i - 1) % (mps[j] - 1)] & masks[j]) >> cumShifts[j]) * (ull)(d_data[i] - 'a' + 1)); //hash =  (ds[j]^(i mod (mps[j] - 1)) % mps[j]) * data[i]
        hash = (hash & mps[j]) + (hash >> shifts[j]);
        hash = hash >= mps[j] ? hash - mps[j] : hash;
        d_a[i] |= hash << cumShifts[j];
    }
    
}
__global__ void FindMatches(int* d_prefixSum, char* d_data, char* d_patterns, int* d_lookupTable, int* d_controlArray, int* d_hashTable, int* d_output) {
    ull j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j + m - 1 <= n) {
        int hash = ((((ull)(d_prefixSum[j + m - 1] - (j ? d_prefixSum[j - 1] : 0)) + q) % q) * (ull)d_lookupTable[(m + ((n - j + q - 2ll) / (q - 1ll)) * (q - 1ll) - n + j) % (q - 1ll)]) % q;
        while (d_controlArray[hash]) {
            int patternIndex = d_hashTable[hash];
            bool match = true;
            for (int i = patternIndex * m, offset = 0; i < patternIndex * m + m; i++, offset++) {
                if (d_patterns[i] != d_data[j + offset]) {
                    match = false;
                    break;
                }
            }
            if (match) {
                d_output[j] = patternIndex;
                return; 
            }
            hash = (hash + 1) % q;
        }
    }
}
__global__ void FindMatchesNew(ull* d_prefixSum, ull* d_lookupTable, int* d_controlArray, int* d_hashTable, int* d_output, ull* d_patternHashes) {
    ull j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j + m - 1 <= n) {
        ull hash = 0;
        for(int k = 0; k < masksz; k++){
            ull tmp = (d_prefixSum[j + m - 1] & masks[k]) >> cumShifts[k];
            if(j){
                tmp = tmp + mps[k] - ((d_prefixSum[j - 1] & masks[k]) >> cumShifts[k]);
                tmp = (tmp >= mps[k] ? tmp - mps[k] : tmp);
            }
            tmp = tmp * (d_lookupTable[(m + ((n - j + mps[k] - 2ll) / (mps[k] - 1ll)) * (mps[k] - 1ll) - n + j) % (mps[k] - 1ll)] >> cumShifts[k]) ;
            tmp = (tmp & mps[k]) + (tmp >> shifts[k]);
            hash |= (tmp >= mps[k] ? tmp - mps[k] : tmp) << cumShifts[k]; 
        }
        
        while (d_controlArray[hash & HTMSK]) {
            if (hash == d_patternHashes[d_hashTable[hash & HTMSK]]) {
                d_output[j] = d_hashTable[hash & HTMSK];
                return;
            }
            hash = (hash == HTMSK) ? 0: hash + 1;
        }
    }
}
class PaperImplementation{
private:
    static int* Step1() {
        int* ret = NULL;
        int *h_lookupTabe = new int [q];
        for (int i = 0, current = 1; i < q; i++, current = (current * d) % q ) 
            h_lookupTabe[i] = current;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&ret, sizeof(int) * q));
        CubDebugExit(hipMemcpy(ret, h_lookupTabe, sizeof(int) * q, hipMemcpyHostToDevice));
        delete[] h_lookupTabe;
        hipDeviceSynchronize();
        return ret;
    }
    static pair<int*, int*> Step2(char * d_patterns) {
        int* d_controlArray = NULL, * d_hashTable = NULL,* h_controlArray = new int[q],* h_hashTable = new int [q];
        for (int i = 0; i < q; i++) {
            h_controlArray[i] = 0;
            h_hashTable[i] = -1;
        }
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_controlArray, sizeof(int) * q));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_hashTable, sizeof(int) * q));
        CubDebugExit(hipMemcpy(d_controlArray, h_controlArray, sizeof(int) * q, hipMemcpyHostToDevice));
        CubDebugExit(hipMemcpy(d_hashTable, h_hashTable, sizeof(int) * q, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        CalculateHashPattern <<< 1, p >>> (d_patterns, d_controlArray, d_hashTable);
        hipDeviceSynchronize();
        delete[] h_hashTable;
        delete[] h_controlArray;
        return { d_controlArray, d_hashTable };
    }
    static int* Step3(char * d_data, int* d_lookupTable) {
        int* d_a = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_a, sizeof(int) * n));
        CalculateHashes << <n / 256, 256 >> > (d_a, d_data, d_lookupTable);
        hipDeviceSynchronize();
        return d_a;
    }
    static int* Step4(int* d_a) {
        int* d_prefixSum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_prefixSum, sizeof(int) * n));
        hipDeviceSynchronize();
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumMod, n));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumMod, n));
        hipFree(d_a);
        hipFree(d_temp_storage);
        return d_prefixSum;
    }
    static int* Step5(int* d_prefixSum, char* d_data, char* d_patterns, int* d_lookupTable, int* d_controlArray, int* d_hashTable) {
        int* h_output = new int [n];
        for (int i = 0; i < n; i++)
            h_output[i] = -1;
        int* d_output = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(int) * n));
        CubDebugExit(hipMemcpy(d_output, h_output, sizeof(int) * n, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        FindMatches << <n / 256, 256 >> > (d_prefixSum, d_data, d_patterns, d_lookupTable, d_controlArray, d_hashTable, d_output);
        CubDebugExit(hipMemcpy(h_output, d_output, sizeof(int) * n, hipMemcpyDeviceToHost));
        hipFree(d_output);
        hipFree(d_prefixSum);
        hipFree(d_data);
        hipFree(d_patterns);
        hipFree(d_lookupTable);
        hipFree(d_controlArray);
        hipFree(d_hashTable);
        return h_output;
    }
public:
    static int* Execute() {
        //1.Load a preprocessed lookup table for di mod q (0 ≤ i ≤ q − 1)
        int* d_lookupTable = Step1();
        //2. Compute the values of h(Pk) for all k (0 ≤ k ≤ p − 1) in parallel and create the hash table HT using the calculated values.
        pair<int*, int*> p = Step2(test.d_patterns);
        //3.Compute the a0, a1,..., an−1 in parallel.
        int* d_a = Step3(test.d_data, d_lookupTable);
        //4.Compute the prefix-sums ˆa0, aˆ1,..., aˆn−1.
        int* d_prefixSum = Step4(d_a);
        //5.  For all j (0 ≤ j ≤ n − m), compute ( ˆaj+m−1 − aˆ j−1) · dm−n−j, which is equal to h(tjtj + 1 ... tj + m−1).If array control[h(tjtj + 1 ... tj + m−1)]  0 then compare the characters of text and pattern with Match(i, j).
        int* h_output = Step5(d_prefixSum, test.d_data, test.d_patterns, d_lookupTable, p.first, p.second);
    
        return h_output;
    }
};
class ProposedImplementation{
private:
    static ull* Step1(){
        ull* ret = NULL;
        ull* h_lookupTabe = new ull [mps[5]];
        ull currents[] = {1, 1, 1, 1, 1, 1};
        for(int i = 0; i < mps[5]; i++){
            h_lookupTabe[i] = 0;
            for(int j = 0; j < masksz; j++){
                h_lookupTabe[i] |= currents[j] << cumShifts[j];
                currents[j] = currents[j] * ds[j];
                currents[j] = (currents[j] & mps[j]) + (currents[j] >> shifts[j]);
                currents[j] = currents[j] >= mps[j] ? currents[j] - mps[j] : currents[j];
            }
            
        }    
        CubDebugExit(g_allocator.DeviceAllocate((void**)&ret, sizeof(ull) * mps[5]));
        CubDebugExit(hipMemcpy(ret, h_lookupTabe, sizeof(ull) * mps[5], hipMemcpyHostToDevice));
        delete[] h_lookupTabe;
        hipDeviceSynchronize();
        return ret;
    }
    static pair<pair<int*, int*>, ull*> Step2(char * d_patterns) {
        int* d_controlArray = NULL, * d_hashTable = NULL,* h_controlArray = new int[HTSZ],* h_hashTable = new int [HTSZ];
        ull * d_patternHashes = NULL;
        for (int i = 0; i < HTSZ; i++) {
            h_controlArray[i] = 0;
            h_hashTable[i] = -1;
        }
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_patternHashes, sizeof(ull) * p));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_controlArray, sizeof(int) * HTSZ));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_hashTable, sizeof(int) * HTSZ));
        CubDebugExit(hipMemcpy(d_controlArray, h_controlArray, sizeof(int) * HTSZ, hipMemcpyHostToDevice));
        CubDebugExit(hipMemcpy(d_hashTable, h_hashTable, sizeof(int) * HTSZ, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        CalculateHashPatternNew <<< 1, p >>> (d_patterns, d_controlArray, d_hashTable, d_patternHashes);
        hipFree(d_patterns);
        hipDeviceSynchronize();
        delete[] h_hashTable;
        delete[] h_controlArray;
        return { {d_controlArray, d_hashTable}, d_patternHashes};
    }
    static ull* Step3(char * d_data, ull* d_lookupTable) {
        ull* d_a = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_a, sizeof(ull) * n));
        CalculateHashesNew << <n / 256, 256 >> > (d_a, d_data, d_lookupTable);
        hipFree(d_data);
        hipDeviceSynchronize();
        return d_a;
    }
    static ull* Step4(ull* d_a) {
        ull* d_prefixSum = NULL;
        void* d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_prefixSum, sizeof(ull) * n));
        hipDeviceSynchronize();
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumModMersennePrime, n));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
        CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumModMersennePrime, n));
        hipFree(d_a);
        hipFree(d_temp_storage);
        return d_prefixSum;
    }
    static int* Step5(ull* d_prefixSum, ull* d_lookupTable, int* d_controlArray, int* d_hashTable, ull* d_patternHashes) {
        int* h_output = new int [n];
        for (int i = 0; i < n; i++)
            h_output[i] = -1;
        int* d_output = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(int) * n));
        CubDebugExit(hipMemcpy(d_output, h_output, sizeof(int) * n, hipMemcpyHostToDevice));
        hipDeviceSynchronize();
        FindMatchesNew << <n / 256, 256 >> > (d_prefixSum, d_lookupTable, d_controlArray, d_hashTable, d_output, d_patternHashes);
        CubDebugExit(hipMemcpy(h_output, d_output, sizeof(int) * n, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        hipFree(d_output);
        hipFree(d_prefixSum);
        hipFree(d_lookupTable);
        hipFree(d_controlArray);
        hipFree(d_hashTable);
        hipFree(d_patternHashes);
        return h_output;
    }
public:
    static int* Execute() {
        CopyDataToDevice();
        ull* d_lookupTable = Step1();
        static pair<pair<int*, int*>, ull*> p = Step2(test.d_patterns);
        ull* d_a = Step3(test.d_data, d_lookupTable);
        ull* d_prefixSum = Step4(d_a);
        int* h_output = Step5(d_prefixSum, d_lookupTable, p.first.first, p.first.second, p.second);
        return h_output;
    }
};
int main(){
    test.Validate(PaperImplementation::Execute());
    return 0;
}

