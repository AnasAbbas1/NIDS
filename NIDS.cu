#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipcub/hipcub.hpp"
#include "cub/device/device_scan.cuh"
#include <stdio.h>
#include <chrono>
#include<iostream>
#include<vector>
#include<algorithm>
#include<string>
#include <fstream>
#include<set>
#include <stdlib.h>
#include <time.h>
#include <random>

using namespace std;
using namespace std::chrono;
using namespace hipcub;
CachingDeviceAllocator g_allocator(true);
const int d = 2;
const long long q = 65521;
const int p = 1024;
const long long n = 1 << 18;
const long long m = 13;
struct testcase {
private:
    string input_str;
    vector<pair<int, int>>expectedMatches;
    char* StringGeneration(int sz) {
        char* ret = new char[sz + 1];
        mt19937 generator{ random_device{}() };
        uniform_int_distribution<int> distribution{ 'a', 'a' + d - 1 };
        string rand_str(sz, '\0');
        for (auto& dis : rand_str)
            dis = distribution(generator);
        for (int i = 0; i < sz; i++) {
            ret[i] = rand_str[i];
        }
        ret[sz] = 0;
        return ret;
    }
    void WriteData(char* data) {
        ofstream myfile;
        myfile.open("outputfiles\\data.txt");
        myfile << string(data);
        myfile.close();
    }
    void WritePatterns(char * patterns) {
        set<string>st;
        ofstream myfile;
        myfile.open("outputfiles\\patterns.txt");
        for (int patternIndex = 0; patternIndex < p; patternIndex++) {
            string pattern = "";
            for (int i = patternIndex * m; i < patternIndex * m + m; i++)
                pattern += h_patterns[i];
            st.insert(pattern);
            myfile << patternIndex << ": " << pattern << endl;
        }
        if (st.size() != p) {
            cout << "Duplicate pattern occurred" << endl; 
        }
        myfile.close();
    }
    void WriteMatches(vector<pair<int, int>> matches, string fileName) {
        ofstream myfile;
        myfile.open(fileName.c_str());
        sort(matches.begin(), matches.end());
        myfile << "Row#\tPattern Index\tposition" << endl;
        for (int i = 0; i < matches.size(); i++) {
            myfile << i << ":\t"<<matches[i].first << "\t\t" << matches[i].second << endl;
        }
        myfile.close();
    }
    char * PatternsGeneration(){
        set<string>st;
        while(st.size() != p){
            char * ptrn = StringGeneration(m);
            st.insert(string(ptrn));
        }
        char * ret = new char[p * m + 1];
        ret[p * m] = 0;
        while(st.size()){
            for(int i = 0; i < m; i++){
                ret[m * (p - st.size()) + i] = (*st.begin())[i];
            }
            st.erase(st.begin());
        }
        return ret;
    }
    void GenerateInputData() {
        h_data = StringGeneration(n);
        d_data = NULL;
        h_patterns = PatternsGeneration();
        d_patterns = NULL;
    }
    void FindPattern(int patternIndex) {
        string pattern = "";
        for (int i = patternIndex * m; i < patternIndex * m + m; i++)
            pattern += h_patterns[i];

        size_t pos = input_str.find(pattern);
        while (pos != string::npos) {
            expectedMatches.push_back({ patternIndex, pos });
            pos = input_str.find(pattern, pos + 1);
        }

    }
    void SolveOnCPU() {
        input_str = string(h_data);
        for (int i = 0; i < p; i++) 
            FindPattern(i);
    }
public:
    char* h_data;
    char* h_patterns;
    char* d_data;
    char* d_patterns;
    testcase() {
        GenerateInputData();
        WriteData(h_data);
        WritePatterns(h_patterns);
        SolveOnCPU();
        WriteMatches(expectedMatches, "outputfiles\\Expected.txt");
    }

    void WriteActual(int* h_output) {
        vector<pair<int, int>>gpuMatches;
        for (int i = 0; i < n; i++) {
            if (h_output[i] != -1) {
                gpuMatches.push_back({h_output[i], i});
            }
        }
        sort(gpuMatches.begin(), gpuMatches.end());
        WriteMatches(gpuMatches, "outputfiles\\Actual.txt");
        bool same = true;
        if (gpuMatches.size() != expectedMatches.size()) {
            cout << "sizes are not equal expected size is " << expectedMatches.size() << " and actual size is " << gpuMatches.size() << endl;
            same = false;
        }
        for (int i = 0;i < min(gpuMatches.size(), expectedMatches.size()); i++) {
            if (gpuMatches[i].first != expectedMatches[i].first || gpuMatches[i].second != expectedMatches[i].second) {
                cout << "Mismatch at position: " << i << endl;
                same = false;
            }
        }
        if (same) {
            cout << "Code works fine" << endl;
        }
        else {
            cout << "Results doesn't match, debug your code" << endl;
        }
        cout << "Enter any key to exit" << endl;
        getchar();
    }

}test;
struct CustomSum
{
    HIPCUB_RUNTIME_FUNCTION __host__ __device__ __forceinline__
        int operator()(const int& a, const int& b) const {
        return (a + b) % q;
    }
}sumMod;
__global__ void CalculateHashPattern(char* d_patterns, int* d_controlArray, int* d_hashTable) {
    int patternIndex = threadIdx.x, patternHash = 0;

    for (int i = patternIndex * m; i < patternIndex * m + m; i++)
        patternHash = (patternHash * d + (d_patterns[i] - 'a' + 1)) % q;

    while (atomicAdd(&d_controlArray[patternHash], 1) != 0)
        patternHash = (patternHash + 1) % q;

    d_hashTable[patternHash] = patternIndex;
}
__global__ void CalculateHashes(int* d_a, char* d_data, int* d_lookupTable) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_a[i] = (d_lookupTable[(n - i - 1) % (q - 1)] * (d_data[i] - 'a' + 1)) % q;
}
__global__ void FindMatches(int* d_prefixSum, char* d_data, char* d_patterns, int* d_lookupTable, int* d_controlArray, int* d_hashTable, int* d_output) {
    long long j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j + m - 1 <= n) {
        int hash = ((((long long)(d_prefixSum[j + m - 1] - (j ? d_prefixSum[j - 1] : 0)) + q) % q) * (long long)d_lookupTable[(m + ((n - j + q - 2ll) / (q - 1ll)) * (q - 1ll) - n + j) % (q - 1ll)]) % q;
        while (d_controlArray[hash]) {
            int patternIndex = d_hashTable[hash];
            bool match = true;
            for (int i = patternIndex * m, offset = 0; i < patternIndex * m + m; i++, offset++) {
                if (d_patterns[i] != d_data[j + offset]) {
                    match = false;
                    break;
                }
            }
            if (match) {
                d_output[j] = patternIndex;
                return; 
            }
            hash = (hash + 1) % q;
        }
    }
}
int* Step1() {
    int* ret = NULL;
    int *h_lookupTabe = new int [q];
    for (int i = 0, current = 1; i < q; i++, current = (current * d) % q ) 
        h_lookupTabe[i] = current;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&ret, sizeof(int) * q));
    CubDebugExit(hipMemcpy(ret, h_lookupTabe, sizeof(int) * q, hipMemcpyHostToDevice));
    delete[] h_lookupTabe;
    hipDeviceSynchronize();
    return ret;
}
pair<int*, int*> Step2(char * d_patterns) {
    int* d_controlArray = NULL, * d_hashTable = NULL,* h_controlArray = new int[q],* h_hashTable = new int [q];
    for (int i = 0; i < q; i++) {
        h_controlArray[i] = 0;
        h_hashTable[i] = -1;
    }
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_controlArray, sizeof(int) * q));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_hashTable, sizeof(int) * q));
    CubDebugExit(hipMemcpy(d_controlArray, h_controlArray, sizeof(int) * q, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_hashTable, h_hashTable, sizeof(int) * q, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    CalculateHashPattern <<< 1, p >>> (d_patterns, d_controlArray, d_hashTable);
    hipDeviceSynchronize();
    delete[] h_hashTable;
    delete[] h_controlArray;
    return { d_controlArray, d_hashTable };
}
int* Step3(char * d_data, int* d_lookupTable) {
    int* d_a = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_a, sizeof(int) * n));
    CalculateHashes << <n / 256, 256 >> > (d_a, d_data, d_lookupTable);
    hipDeviceSynchronize();
    return d_a;
}
int* Step4(int* d_a) {
    int* d_prefixSum = NULL;
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_prefixSum, sizeof(int) * n));
    hipDeviceSynchronize();
    CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumMod, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    CubDebugExit(DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_a, d_prefixSum, sumMod, n));
    hipFree(d_a);
    hipFree(d_temp_storage);
    return d_prefixSum;
}
int* Step5(int* d_prefixSum, char* d_data, char* d_patterns, int* d_lookupTable, int* d_controlArray, int* d_hashTable) {
    int* h_output = new int [n];
    for (int i = 0; i < n; i++)
        h_output[i] = -1;
    int* d_output = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(int) * n));
    CubDebugExit(hipMemcpy(d_output, h_output, sizeof(int) * n, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    FindMatches << <n / 256, 256 >> > (d_prefixSum, d_data, d_patterns, d_lookupTable, d_controlArray, d_hashTable, d_output);
    CubDebugExit(hipMemcpy(h_output, d_output, sizeof(int) * n, hipMemcpyDeviceToHost));
    hipFree(d_output);
    hipFree(d_prefixSum);
    hipFree(d_data);
    hipFree(d_patterns);
    hipFree(d_lookupTable);
    hipFree(d_controlArray);
    hipFree(d_hashTable);
    return h_output;
}
int* PaperImplementation() {
    //1.Load a preprocessed lookup table for di mod q (0 ≤ i ≤ q − 1)
    int* d_lookupTable = Step1();
    //2. Compute the values of h(Pk) for all k (0 ≤ k ≤ p − 1) in parallel and create the hash table HT using the calculated values.
    pair<int*, int*> p = Step2(test.d_patterns);
    //3.Compute the a0, a1,..., an−1 in parallel.
    int* d_a = Step3(test.d_data, d_lookupTable);
    //4.Compute the prefix-sums ˆa0, aˆ1,..., aˆn−1.
    int* d_prefixSum = Step4(d_a);
    //5.  For all j (0 ≤ j ≤ n − m), compute ( ˆaj+m−1 − aˆ j−1) · dm−n−j, which is equal to h(tjtj + 1 ... tj + m−1).If array control[h(tjtj + 1 ... tj + m−1)]  0 then compare the characters of text and pattern with Match(i, j).
    int* h_output = Step5(d_prefixSum, test.d_data, test.d_patterns, d_lookupTable, p.first, p.second);

    return h_output;
}
void CopyDataToDevice() {
    CubDebugExit(g_allocator.DeviceAllocate((void**)&test.d_data, sizeof(char) * n));
    CubDebugExit(hipMemcpy(test.d_data, test.h_data, sizeof(char) * n, hipMemcpyHostToDevice));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&test.d_patterns, sizeof(char) * p * m));
    CubDebugExit(hipMemcpy(test.d_patterns, test.h_patterns, sizeof(char) * p * m, hipMemcpyHostToDevice));
    delete[] test.h_patterns;
    delete[] test.h_data;
}
int main(){
    CopyDataToDevice();
    test.WriteActual(PaperImplementation());
    return 0;
}

